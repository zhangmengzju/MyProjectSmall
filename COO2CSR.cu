#include <hip/hip_runtime.h>
#include <iostream>
#include "hipsparse.h"
using namespace std;
   
int SpasrseCOO2CSR(
		const int *h_cooRowInd,
		int nnz, 
		int m, 
		int *h_csrRowPtr
		)
{
	////start1////
	hipsparseHandle_t handle = 0;
	hipsparseStatus_t cusparseStatus = hipsparseCreate(&handle);
	////end1////

	////start2////
	int *d_cooRowInd;
	hipMalloc((void**)&d_cooRowInd, sizeof(int)*(nnz));
	hipMemcpy(d_cooRowInd, h_cooRowInd, (nnz)*sizeof(int), hipMemcpyHostToDevice);

	int *d_csrRowPtr;
	hipMalloc((void**)&d_csrRowPtr, sizeof(int)*(m+1));
	hipMemcpy(d_csrRowPtr, h_csrRowPtr, (m+1)*sizeof(int), hipMemcpyHostToDevice);
	////end2////
		
	hipsparseXcoo2csr(handle, 	//cusparseHandle_t
					d_cooRowInd,
					nnz, 
					m, 
					d_csrRowPtr, 
					HIPSPARSE_INDEX_BASE_ZERO);	//cusparseIndexBase_t			
					 
	////start3//// 
	hipMemcpy(h_csrRowPtr, d_csrRowPtr, (m+1)*sizeof(int), hipMemcpyDeviceToHost); 
	hipsparseDestroy(handle);	 
	hipFree(d_cooRowInd); 	 
	hipFree(d_csrRowPtr); 
	hipDeviceReset();
	////end3////
	
	return 1;
}

int main()
{
	float* cooVal = new float[9];
    int* cooCol = new int[9]; 	
    int* cooRowInd = new int[9];

    float* pCooVal = cooVal;
    int* pCooCol = cooCol;	
    int* pCooRowInd = cooRowInd;

    *pCooVal = 1.0F; pCooVal++;
    *pCooVal = 4.0F; pCooVal++;
    *pCooVal = 2.0F; pCooVal++;
	*pCooVal = 3.0F; pCooVal++;
	*pCooVal = 5.0F; pCooVal++;
	*pCooVal = 7.0F; pCooVal++;
	*pCooVal = 8.0F; pCooVal++;
	*pCooVal = 9.0F; pCooVal++;
	*pCooVal = 6.0F; pCooVal++;

	*pCooRowInd = 0; pCooRowInd++;
	*pCooRowInd = 0; pCooRowInd++;
	*pCooRowInd = 1; pCooRowInd++;
	*pCooRowInd = 1; pCooRowInd++;
	*pCooRowInd = 2; pCooRowInd++;
	*pCooRowInd = 2; pCooRowInd++;
	*pCooRowInd = 2; pCooRowInd++;
	*pCooRowInd = 3; pCooRowInd++;
	*pCooRowInd = 3; pCooRowInd++;

	*pCooCol = 0; pCooCol++;
	*pCooCol = 1; pCooCol++;
	*pCooCol = 1; pCooCol++;
	*pCooCol = 2; pCooCol++;
	*pCooCol = 0; pCooCol++;
	*pCooCol = 3; pCooCol++;
	*pCooCol = 4; pCooCol++;
	*pCooCol = 2; pCooCol++;
	*pCooCol = 4; pCooCol++;
	//以上代码在给矩阵A赋值，使用COO格式
	
	int *csrRowPtr = new int[5];//m+1	

	int c = SpasrseCOO2CSR(
		cooRowInd,//const int *h_cooRowInd
		9,//int nnz,
		4,//int m,		
		csrRowPtr//int *h_csrRowPtr
	   );
	   
	for(int i=0; i<5; i++)
		cout<< csrRowPtr[i]<<endl;	   
	cout<< "COO2CSR Result" << c << endl;  
	   
	return 0;
} 
