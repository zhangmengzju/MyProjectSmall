#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "hipsparse.h"
#include <mpi.h>
#include <vector>
#include <fstream>
#include <sstream>//在string后连接int等类型
#include <string> 
#include <stdlib.h>//使能在g++中编译通过atoi
//#include <io.h>//windows下得到目录下的文件信息

#include <dirent.h>//linux下得到目录下的文件信息opendir
#include <sys/stat.h>//linux下得到目录下的文件信息
#include <sys/types.h>//linux创建目录mkdir

#include <algorithm>//sort
#include <iomanip>//cout setprecision
#include <map>
#include <time.h>
#include <errno.h>

using namespace std;//g++中要对string、vector、ifstream、stringstream、endl、cout等显式添加std:: 

/////////////////////class COO begin///////////////////////////////////////////////
class COO {
	public:
		vector<int> coo_rows;//避免像使用数组时，需要先确定大小
		vector<int> coo_cols;
		vector<float> coo_vals;
		int coo_rows_max ;//m=coo_rows_max+1 coo_rows下标的最大值，若下标最大为3，则实际有4行
		int coo_cols_max ;//n=coo_cols_max+1 coo_cols下标的最大值，若下标最大为3，则实际有4列
		
		COO(int default_max) {
			coo_rows_max = default_max;
			coo_cols_max = default_max;
		}
};
/////////////////////class COO end///////////////////////////////////////////////


/////////////////////class CSR begin///////////////////////////////////////////////
class CSR {
	public:
		vector<int> csr_row_ptrs;//避免像使用数组时，需要先确定大小
		vector<int> csr_cols;
		vector<float> csr_vals;
		int csr_rows_max ;//m=csr_rows_max+1 csr_rows下标的最大值，若下标最大为3，则实际有4行
		int csr_cols_max ;//n=csr_cols_max+1 csr_cols下标的最大值，若下标最大为3，则实际有4列
		
		CSR( int default_max) {
			csr_rows_max = default_max;
			csr_cols_max = default_max;
		}


	//	!!!!!!!!!!!!!!!!!!!!!!
    /////////////////LongID与IntID的映射表
	//	map<long,int> Long2IntMap;
    //	vector<long> Int2LongVector;
    //	!!!!!!!!!!!!!!!!!!!!!!

};
/////////////////////class CSR end///////////////////////////////////////////////



/////////////////////class CSC begin///////////////////////////////////////////////
class CSC {
        public:
                vector<int> csc_col_ptrs;//避免像使用数组时，需要先确定大小
                vector<int> csc_rows;
                vector<float> csc_vals;
                int csc_rows_max ;//m=csc_rows_max+1 csc_rows下标的最大值，若下标最大为3，则实际有4行
                int csc_cols_max ;//n=csc_cols_max+1 csc_cols下标的最大值，若下标最大为3，则实际有4列

                CSC( int default_max) {
                        csc_rows_max = default_max;
                        csc_cols_max = default_max;
                }
		
		CSR use_CSC_Create_CSR() {
			CSR csr(-1);
                        csr.csr_rows_max = csc_rows_max;
                        csr.csr_cols_max = csc_cols_max;

			csr.csr_row_ptrs = csc_col_ptrs;
			csr.csr_cols = csc_rows;
			csr.csr_vals = csc_vals;

			return csr;
		}
};
/////////////////////class CSC end///////////////////////////////////////////////


/*////////////////////////////////////////////////////////////////////////////////
//////////////////saveCSRAsFiles begin////////////////////////////////////////////////
void saveCSRAsFiles(string fileNameDir, CSR csr) {

        //目录不为空则删除，然后新建目录
        if(NULL!=opendir(fileNameDir.c_str())){
                string cmd = "rm -rf ";
                cmd += fileNameDir.c_str();
                cout << "[cmd]" <<cmd.c_str() <<endl;
                system( cmd.c_str() );
        }
        mkdir(fileNameDir.c_str(), 0775);


	/////////save file_csr_row_ptrs_and_rows_max_and_cols_max
        ostringstream oss_1;//在string后连接int等类型
        oss_1 << fileNameDir << "//row_max_and_col_max_and_row_ptrs.txt";
        string fileNameTmp_1 = oss_1.str();

        ofstream fout_1(fileNameTmp_1.c_str());

	//save csr_rows_max
	fout_1 << csr.csr_rows_max <<endl;

 	//save csr_cols_max	
	fout_1 << csr.csr_rows_max <<endl; 

 	//save csr_row_ptrs
        for (int k= 0; k < csr.csr_row_ptrs.size(); k++){
		
		////////////////////////////////////
		!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
		要保存原始的LongID，而非映射后的IntID
		///////////////////////////////////

		cout << csr.csr_row_ptrs.get(k) << endl;
                fout_1 << csr.csr_row_ptrs.get(k) << endl;
        }
        fout_1.close();




 	/////////save file_csr_cols_and_csr_vals
        ostringstream oss_2;//在string后连接int等类型
        oss_2 << fileNameDir << "//cols_and_vals";
        string fileNameTmp_2 = oss_2.str();

        ofstream fout_2(fileNameTmp_1.c_str());

        for (int k= 0; k < csr.csr_cols.size(); k++){
                cout << csr.csr_cols.get(k) << "," << csr.csr_vals.get(k)<<endl;
                fout_2 << csr.csr_cols.get(k) << "," << csr.csr_vals.get(k)<<endl;

        }
        fout_2.close();


	!!!!!!!!!!!!!!!!!!!!!!
	/////////////////save LongID与IntID的映射表
	!!!!!!!!!!!!!!!!!!!!!!

}
//////////////////saveCSRAsFiles end//////////////////////////////////////////////////



//////////////////CSRFileReader begin///////////////////////////////////////////
CSR CSRFileReader(string file_csr_row_ptrs_and_rows_max_and_cols_max, 
		  string file_csr_cols_and_csr_vals,		   
		  const char regex)
{
        CSR csr(-1);

        
        map<long,int> Long2IntMap;
        vector<long> Int2LongVector;




	//这里从文件中加载csr_row_ptrs和csr_rows_max和csr_cols_max
        ifstream fin_1(file_csr_cols_and_csr_vals.c_str());//在g++中使用字符串str时，要用str.c_str()
        string line_1;
        int ind = 0;
        while (getline(fin_1, line_1))
        {
                //cout << "Read from file: " << line << endl; 
                long row = atol(line_1.c_str());//string->long
                if(ind==0)
                        csr.csr_rows_max = row;
                else if(ind==1)
                        csr.csr_cols_max = row;
                else{
                        int new_row = LongID_To_IntID(Long2IntMap, Int2LongVector, row);
                        cout<< "[" << row << "][" << new_row << "]" << endl;
                        csr.csr_rows.push_back(new_row);
                }
                ind++;
        }
        fin_1.close();



	//这里从文件中加载csr_cols和csr_vals两个vector
	ifstream fin_2(file_csr_cols_and_csr_vals.c_str());//在g++中使用字符串str时，要用str.c_str()
        string line_2;
        while (getline(fin_2, line_2))
        {
                vector <string> fields = split(line_2, regex);

                if (fields.size() == 2 ) {
         		int col = atoi(fields[0].c_str());//string->int
                        float val = atof(fields[1].c_str());//string->float

                        csr.csr_cols.push_back(col);
                        csr.csr_vals.push_back(val);
                }
        }
	fin_2.close();


	/////
        int* csr_rows_arr = &csr.csr_rows[0];//vector转成array
        int* csr_cols_arr = &csr.csr_cols[0];
        float* csr_vals_arr = &csr.csr_vals[0];

        //for (int i = 0; i < coo.coo_rows.size(); i++)
        //      cout << coo_rows_arr[i] << endl;
        //for (int i = 0; i < coo.coo_cols.size(); i++)
        //      cout << coo_cols_arr[i] << endl;
        //for (int i = 0; i < coo.coo_vals.size(); i++)
        //      cout << coo_vals_arr[i] << endl; 

        cout << "csr_rows_max " << csr.csr_rows_max << endl;
        cout << "csr_cols_max " << csr.csr_cols_max << endl;

        return csr;
}


CSR useCSRFileReader(string file_csr_row_ptrs_and_rows_max_and_cols_max,
                     string file_csr_cols_and_csr_vals, 
	             const char regex)
{
        return CSRFileReader(file_csr_row_ptrs_and_rows_max_and_cols_max,
                             file_csr_cols_and_csr_vals, 
                             regex);
}


//////////////////CSRFileReader end/////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////*/






//////////////////COOFileReader begin//////////////////////////////////////////////////////////////////////////////
vector<string> &split(const string &s, char delim, vector<string> &elems) {
	stringstream ss(s);
	string item;
	while (getline(ss, item, delim)) {
		elems.push_back(item);
	}
	return elems;
}

vector<string> split(const string &s, char delim) {
	vector<string> elems;
	split(s, delim, elems);
	return elems;
}


int LongID_To_IntID(	map<long,int>& Long2IntMap,
			vector<long>& Int2LongVector , 
			long oriID)
{
	int nextLoc = Long2IntMap.size();
	if(Long2IntMap.find(oriID) == Long2IntMap.end())//将以前没有出现过的long型的原始ID，加入到HashMap中
	{
		Long2IntMap[oriID] = nextLoc ;
		Int2LongVector.push_back(oriID);
	}
	
	return Long2IntMap[oriID];
}


COO COOFileReader(string filename, const char regex)
{
	COO coo(-1);

	ifstream fin(filename.c_str());//在g++中使用字符串str时，要用str.c_str()


        map<long,int> Long2IntMap;
        vector<long> Int2LongVector;

	
	string line;
	while (getline(fin, line))
	{ 
		//cout << "Read from file: " << line << endl; 
		vector <string> fields = split(line, regex);

		if (fields.size() == 3 ) {
			long row = atol(fields[0].c_str());//string->long
			int col = atoi(fields[1].c_str());//string->int
			float val = atof(fields[2].c_str());//string->float
			
			int new_row = LongID_To_IntID(Long2IntMap, Int2LongVector, row);
			//cout<< "[" << row << "][" << new_row << "][" << col << "][" << setprecision(20) << val << "]" << endl;	


			coo.coo_rows.push_back(new_row);
			coo.coo_cols.push_back(col);
			coo.coo_vals.push_back(val);

			coo.coo_rows_max = new_row > coo.coo_rows_max ? new_row : coo.coo_rows_max;
			coo.coo_cols_max = col > coo.coo_cols_max ? col : coo.coo_cols_max; 		
		}
	}

	int* coo_rows_arr = &coo.coo_rows[0];//vector转成array
	int* coo_cols_arr = &coo.coo_cols[0];
	float* coo_vals_arr = &coo.coo_vals[0];

	//for (int i = 0; i < coo.coo_rows.size(); i++)
	//	cout << coo_rows_arr[i] << endl;
	//for (int i = 0; i < coo.coo_cols.size(); i++)
	//	cout << coo_cols_arr[i] << endl;
	//for (int i = 0; i < coo.coo_vals.size(); i++)
	//	cout << coo_vals_arr[i] << endl; 
		
	cout << "coo_rows_max " << coo.coo_rows_max << endl;
	cout << "coo_cols_max " << coo.coo_cols_max << endl;	
	
	fin.close();////////////
	return coo;
}

COO useCOOFileReader(string filename, const char regex)
{
	return COOFileReader(filename, regex);	
}
//////////////////COOFileReader end//////////////////////////////////////////////////////////////////////////////


//////////////////SpasrseCOO2CSR begin//////////////////////////////////////////////////////////////////////////////
int * SpasrseCOO2CSR(
		const int *h_cooRowInd,
		int nnz, 
		int m, 
		int *h_csrRowPtr
		)
{
	////start1////
	hipsparseHandle_t handle = 0;
	hipsparseStatus_t cusparseStatus = hipsparseCreate(&handle);
	////end1////

	////start2////
	int *d_cooRowInd;
	hipMalloc((void**)&d_cooRowInd, sizeof(int)*(nnz));
	hipMemcpy(d_cooRowInd, h_cooRowInd, (nnz)*sizeof(int), hipMemcpyHostToDevice);

	int *d_csrRowPtr;
	hipMalloc((void**)&d_csrRowPtr, sizeof(int)*(m+1));
	hipMemcpy(d_csrRowPtr, h_csrRowPtr, (m+1)*sizeof(int), hipMemcpyHostToDevice);
	////end2////
		
	hipsparseXcoo2csr(handle, 	//hipsparseHandle_t
					d_cooRowInd,
					nnz, 
					m, 
					d_csrRowPtr, 
					HIPSPARSE_INDEX_BASE_ZERO);	//hipsparseIndexBase_t			
					 
	////start3//// 
	hipMemcpy(h_csrRowPtr, d_csrRowPtr, (m+1)*sizeof(int), hipMemcpyDeviceToHost); 
	hipsparseDestroy(handle);	 
	hipFree(d_cooRowInd); 	 
	hipFree(d_csrRowPtr); 
	hipDeviceReset();
	////end3////
	
	return h_csrRowPtr;
}

CSR useSparseCOO2CSR(COO coo){

	CSR csr(-1);
	csr.csr_cols = coo.coo_cols;
	csr.csr_vals = coo.coo_vals;	
	csr.csr_rows_max = coo.coo_rows_max;//coo_rows_max为下标的最大值，若下标最大为3，则实际有4行m=coo_rows_max+1
	csr.csr_cols_max = coo.coo_cols_max;	
	
	//csr_row_ptrs的vector中共m+1=coo_rows_max+2个元素
	for(int i=0; i<=coo.coo_rows_max+1; i++)//这里先创建出一个连续存放了coo_rows_max+2个元素的vector,然后将vector转成array
		csr.csr_row_ptrs.push_back(-1);
	
	int * csrRowPtr = SpasrseCOO2CSR(&coo.coo_rows[0], csr.csr_vals.size(), csr.csr_rows_max+1, &csr.csr_row_ptrs[0]);
	//for(int i=0 ;i<=coo.coo_rows_max+1; i++ ) {
	//	cout << csr.csr_row_ptrs[i] << endl;		
	//}
	cout<<"csrRowPtr.size" << csr.csr_row_ptrs.size() <<endl;
	return csr;
}
//////////////////SpasrseCOO2CSR end//////////////////////////////////////////////////////////////////////////////



//////////////////SpasrseCSR2CSC begin//////////////////////////////////////////////////////////////////////////////
CSC SpasrseCSR2CSC( CSR csr )
{
  	int m = csr.csr_rows_max + 1;
        int n = csr.csr_cols_max + 1;
	int nnz = csr.csr_vals.size();

	//////CSC init start///////////////////////
	CSC csc(-1);
        csc.csc_rows_max = csr.csr_cols_max;//coo_rows_max为下标的最大值，若下标最大为3，则实际有4行m=coo_rows_max+1
        csc.csc_cols_max = csr.csr_rows_max;
	 
        for(int i=0; i<= n; i++)//csc_col_ptrs的vector中共n+1个元素
                csc.csc_col_ptrs.push_back(-1);//这里先创建出一个连续存放了n+1个元素的vector,然后将vector转成array
	for(int i=0; i < nnz; i++)
		csc.csc_rows.push_back(-1);
	for(int i=0; i < nnz; i++)
                csc.csc_vals.push_back(-1);
	//////CSC init end/////////////////////////


        ////start1////
        hipsparseHandle_t handle = 0;
        hipsparseStatus_t cusparseStatus = hipsparseCreate(&handle);
        ////end1////


        ////start2////
        /////csr
	int *h_csrRowPtrs = &csr.csr_row_ptrs[0];
	int *d_csrRowPtrs;
        hipMalloc((void**)&d_csrRowPtrs, sizeof(int)*(m+1));
        hipMemcpy(d_csrRowPtrs, h_csrRowPtrs, (m+1)*sizeof(int), hipMemcpyHostToDevice);

	int *h_csrCols = &csr.csr_cols[0];
        int *d_csrCols;
        hipMalloc((void**)&d_csrCols, sizeof(int)*(nnz));
        hipMemcpy(d_csrCols, h_csrCols, (nnz)*sizeof(int), hipMemcpyHostToDevice);
        
	float *h_csrVals = &csr.csr_vals[0];
	float *d_csrVals;
        hipMalloc((void**)&d_csrVals, sizeof(float)*(nnz));
        hipMemcpy(d_csrVals, h_csrVals, (nnz)*sizeof(float), hipMemcpyHostToDevice);

	/////csc
	int *h_cscColPtrs = &csc.csc_col_ptrs[0];
	int *d_cscColPtrs;
        hipMalloc((void**)&d_cscColPtrs, sizeof(int)*(n+1));
        hipMemcpy(d_cscColPtrs, h_cscColPtrs, (n+1)*sizeof(int), hipMemcpyHostToDevice);

	int *h_cscRows = &csc.csc_rows[0]; 
	int *d_cscRows;
        hipMalloc((void**)&d_cscRows, sizeof(int)*(nnz));
        hipMemcpy(d_cscRows, h_cscRows, (nnz)*sizeof(int), hipMemcpyHostToDevice);

	float *h_cscVals = &csc.csc_vals[0];
	float *d_cscVals;
        hipMalloc((void**)&d_cscVals, sizeof(float)*(nnz));
        hipMemcpy(d_cscVals, h_cscVals, (nnz)*sizeof(float), hipMemcpyHostToDevice);
	////end2////


        hipsparseScsr2csc(
			handle,//handle
			m,//m
			n,//n
			nnz,//nnz

			d_csrVals,//csrval
			d_csrRowPtrs,//csrRowPtr
			d_csrCols,//csrColInd

			d_cscVals,//cscVal
			d_cscRows,//cscRowInd
			d_cscColPtrs,//cscColPtr

			HIPSPARSE_ACTION_NUMERIC,//copyValues, HIPSPARSE_ACTION_NUMERIC: operation performed on data and indices
 			HIPSPARSE_INDEX_BASE_ZERO//idxBase
                        );             

        ////start3//// 
        hipMemcpy(h_cscColPtrs, d_cscColPtrs, (n+1)*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_cscRows, d_cscRows, (nnz)*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_cscVals, d_cscVals, (nnz)*sizeof(float), hipMemcpyDeviceToHost);

        hipsparseDestroy(handle);

	hipFree(d_csrRowPtrs);
        hipFree(d_csrCols);
 	hipFree(d_csrVals);

        hipFree(d_cscColPtrs);        
	hipFree(d_cscRows);
        hipFree(d_cscVals);

        hipDeviceReset();
        ////end3////

        return csc;
}

CSC useSparseCSR2CSC(CSR csr){

       	cout << "+++++++++++++++++++++++" << endl;
  	cout << "csr_row_ptrs.size "<< csr.csr_row_ptrs.size() << endl;
        //for(int i=0; i<csr.csr_row_ptrs.size(); i++ ) 
        //     cout << csr.csr_row_ptrs[i] << endl;       

        cout << "csr_cols.size " << csr.csr_cols.size() << endl;
        //for(int i=0; i<csr.csr_cols.size(); i++ )    
        //      cout << csr.csr_cols[i] << endl;    

        cout << "csr_vals.size " << csr.csr_vals.size() << endl;
        //for(int i=0; i<csr.csr_vals.size(); i++ )    
        //      cout << csr.csr_vals[i] << endl; 
	cout << "-----------------------" << endl;


        CSC csc = SpasrseCSR2CSC(csr);
        

	cout << "+++++++++++++++++++++++" << endl;
	cout << "csc_col_ptrs.size " << csc.csc_col_ptrs.size() << endl;
	//for(int i=0; i<csc.csc_col_ptrs.size(); i++ ) 
        //      cout << csc.csc_col_ptrs[i] << endl;          

	cout << "csc_rows.size " << csc.csc_rows.size() << endl;
        //for(int i=0; i<csc.csc_rows.size(); i++ ) 
        //      cout << csc.csc_rows[i] << endl;       

	cout << "csc_vals.size " << csc.csc_vals.size() << endl;
        //for(int i=0; i<csc.csc_vals.size(); i++ ) 
        //      cout << csc.csc_vals[i] << endl;       
	cout << "----------------------" << endl;

        return csc;
}
//////////////////SpasrseCSR2CSC end//////////////////////////////////////////////////////////////////////////////




//////////////////SparseMultiply begin//////////////////////////////////////////////////////////////////////////////
CSR SparseMultiply(int m,int n, int k,int nnzA,int nnzB,
					float *h_A,int *h_RowA,int *h_ColA,
					float *h_B,int *h_RowB,int *h_ColB)
{
	int baseC,nnzC;
	// nnzTotalDevHostPtr points to host memory
	int *nnzTotalDevHostPtr = &nnzC;

	hipsparseHandle_t handle=0;
	hipsparseStatus_t cusparseStatus;

	hipsparseMatDescr_t descrA=0;
	hipsparseMatDescr_t descrB=0;
	hipsparseMatDescr_t descrC=0;

	float *d_A;
	int *d_RowA;
	int *d_ColA;

	float *d_B;
	int *d_RowB;
	int *d_ColB;
	
	float *d_C;
	int *d_RowC;
	int *d_ColC;


	cusparseStatus = hipsparseCreate(&handle);

	hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

	hipMalloc((void**)&d_A, sizeof(float)*(nnzA));
	hipMalloc((void**)&d_RowA, sizeof(int)*(m+1));
	hipMalloc((void**)&d_ColA, sizeof(int)*(nnzA));

	hipMalloc((void**)&d_B, sizeof(float)*(nnzB));
	hipMalloc((void**)&d_RowB, sizeof(int)*(n+1));
	hipMalloc((void**)&d_ColB, sizeof(int)*(nnzB));


	hipMemcpy(d_A, h_A, nnzA*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_RowA, h_RowA, (m+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_ColA, h_ColA, nnzA*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(d_B, h_B, nnzB*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_RowB, h_RowB, (n+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_ColB, h_ColB, nnzB*sizeof(int), hipMemcpyHostToDevice);

	cusparseStatus = hipsparseCreateMatDescr(&descrA);
	hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO);

	cusparseStatus = hipsparseCreateMatDescr(&descrB);
	hipsparseSetMatType(descrB,HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatIndexBase(descrB,HIPSPARSE_INDEX_BASE_ZERO);

	cusparseStatus = hipsparseCreateMatDescr(&descrC);
	hipsparseSetMatType(descrC,HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatIndexBase(descrC,HIPSPARSE_INDEX_BASE_ZERO);
	//////////////////////////////////////////////////////////////////////////
	hipMalloc((void**)&d_RowC, sizeof(int)*(m+1));

	hipsparseXcsrgemmNnz(
		handle,
		HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		m,
		n,
		k,
		descrA,
		nnzA,
		d_RowA,
		d_ColA,
		descrB,
		nnzB,
		d_RowB,
		d_ColB,
		descrC,
		d_RowC,
		nnzTotalDevHostPtr);

	if (NULL != nnzTotalDevHostPtr)
	{	
		cout << "[null != nnzTotalDevHostPtr]" << endl;
		nnzC = *nnzTotalDevHostPtr;
	} else {
		hipMemcpy(&nnzC, d_RowC+m, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&baseC, d_RowC, sizeof(int), hipMemcpyDeviceToHost);

		cout << "[null == nnzTotalDevHostPtr]" << endl;
		cout << "[null == nnzTotalDevHostPtr nnzC ]" << nnzC << endl;
		cout << "[null == nnzTotalDevHostPtr baseC ]" << baseC << endl;
		nnzC -= baseC;
	}

	
	//////////////////////////////////////////////////////
	//////////////////////////////////////////////////////
	cout << "[nnzC First]" << nnzC << endl;

	vector<int> c_csr_row_ptrs;
	for(int i=0; i<=m; i++)//这里先创建出一个连续存放了m+1个元素的vector,然后将vector转成array
		c_csr_row_ptrs.push_back(-1);
		
	vector<int> c_cols;	
	vector<float> c_vals;
	for(int i=0; i<nnzC; i++) { //这里先创建出一个连续存放了nnzC个元素的vector,然后将vector转成array
		c_cols.push_back(-1);
		c_vals.push_back(-1);
	}		
	///host上的矩阵相乘后的结果
	float *h_C = &c_vals[0];
	int *h_RowC = &c_csr_row_ptrs[0];
	int *h_ColC = &c_cols[0];	
	//////////////////////////////////////////////////////
	//////////////////////////////////////////////////////
	
	
	
	hipMalloc((void**)&d_ColC, sizeof(int)*nnzC);
	hipMalloc((void**)&d_C, sizeof(float)*nnzC);

	
	hipsparseScsrgemm(
		handle, 
		HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		m, 
		n, 
		k,
		descrA, 
		nnzA,
		d_A, 
		d_RowA, 
		d_ColA,
		descrB,
		nnzB,
		d_B, 
		d_RowB, 
		d_ColB,
		descrC,
		d_C, 
		d_RowC, 
		d_ColC);


	hipMemcpy(h_C, d_C, nnzC*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_RowC, d_RowC, (m+1)*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_ColC, d_ColC, nnzC*sizeof(int), hipMemcpyDeviceToHost);

	hipsparseDestroy(handle);

	hipFree(d_A);
	hipFree(d_RowA);
	hipFree(d_ColA);

	hipFree(d_B);
	hipFree(d_RowB);
	hipFree(d_ColB);

	hipFree(d_C);
	hipFree(d_RowC);
	hipFree(d_ColC);

	hipDeviceReset();
	
	//////////////////////////////////////////////////////
	//////////////////////////////////////////////////////	
	CSR csr_C(-1);
	csr_C.csr_rows_max = m-1;//csr_rows下标的最大值，若下标最大为3，则实际有4行
	csr_C.csr_cols_max = k-1;//csr_cols下标的最大值，若下标最大为3，则实际有4列
		
	cout << "[nnzC]" << nnzC << endl;	
	for(int i=0; i<=m; i++) {
	//	cout << "[h_RowC]"<< h_RowC[i] << endl; 
		csr_C.csr_row_ptrs.push_back(h_RowC[i]);	
	}
	for(int i=0; i<nnzC; i++) {
	//	cout << "[h_ColC]"<< h_ColC[i] << endl;
		csr_C.csr_cols.push_back(h_ColC[i]);	
	}
	for(int i=0; i<nnzC; i++) {
	//	cout << "[h_C]"<< h_C[i] << endl;
		csr_C.csr_vals.push_back(h_C[i]);
	}
	//////////////////////////////////////////////////////
	//////////////////////////////////////////////////////
	
	return csr_C;
}

CSR useSparseMultiply(CSR csr_A, CSR csr_B){

	float* ValA = &csr_A.csr_vals[0];
    	int* RowA = &csr_A.csr_row_ptrs[0];
    	int* ColA = &csr_A.csr_cols[0];     
	//以上代码在给矩阵A赋值，使用CSR格式
	
	float* ValB = &csr_B.csr_vals[0];
	int* RowB = &csr_B.csr_row_ptrs[0];
	int* ColB = &csr_B.csr_cols[0]; 
	//以上代码在给矩阵B赋值，使用CSR格式 
	

	int n_new = (csr_A.csr_cols_max+1) >  (csr_B.csr_rows_max+1) ?  (csr_A.csr_cols_max+1) : (csr_B.csr_rows_max+1) ;

	CSR csr_C = SparseMultiply(
		csr_A.csr_rows_max+1,//int m,
		n_new,		     //int n,
		csr_B.csr_cols_max+1,//int k,
		csr_A.csr_vals.size(),//int nnzA,
		csr_B.csr_vals.size(),//int nnzB,
		
		ValA,//float *h_A,
		RowA,//int *h_RowA,
		ColA,//int *h_ColA,
		
		ValB,//float *h_B,
		RowB,//int *h_RowB,
		ColB //int *h_ColB
	   );
	   
	   
	   	
	cout << "+++1++++" << endl;
	//for(int i=0; i<= csr_C.csr_rows_max+1; i++) //行数m = 下标最大值csr_rows_max+1
	//	cout << csr_C.csr_row_ptrs[i] << endl;
 	cout << "multiply csr_c.csr_row_ptrs.size " << csr_C.csr_row_ptrs.size() << endl;	

	cout << "+++2++++" << endl;
	//for(int i=0; i< csr_C.csr_cols.size(); i++) 
	//	cout << csr_C.csr_cols[i] << endl;
 	cout <<"mulitiply csr_c.csr_cols.size " << csr_C.csr_cols.size() << endl;	

	cout << "+++3++++" << endl;
	//for(int i=0; i< csr_C.csr_vals.size(); i++) 
	//	cout << csr_C.csr_vals[i] << endl;    
 	cout<< "multiply csr_c.csr_vals.size " << csr_C.csr_vals.size() << endl;
	return csr_C;
}
//////////////////SparseMultiply end//////////////////////////////////////////////////////////////////////////////




//////////////////SparseAddition start///////////////////////////////////////////////////////////////////////////
CSR SparseAddition(int m,int n, int nnzA,int nnzB,
                                        float *h_A,int *h_RowA,int *h_ColA,
                                        float *h_B,int *h_RowB,int *h_ColB)
{
        int baseC,nnzC;
        // nnzTotalDevHostPtr points to host memory
        int *nnzTotalDevHostPtr = &nnzC;

        hipsparseHandle_t handle=0;
        hipsparseStatus_t cusparseStatus;

        hipsparseMatDescr_t descrA=0;
        hipsparseMatDescr_t descrB=0;
        hipsparseMatDescr_t descrC=0;

        float *d_A;
        int *d_RowA;
        int *d_ColA;

        float *d_B;
        int *d_RowB;
        int *d_ColB;

        float *d_C;
        int *d_RowC;
        int *d_ColC;


        cusparseStatus = hipsparseCreate(&handle);

        hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

        hipMalloc((void**)&d_A, sizeof(float)*(nnzA));
        hipMalloc((void**)&d_RowA, sizeof(int)*(m+1));
        hipMalloc((void**)&d_ColA, sizeof(int)*(nnzA));

        hipMalloc((void**)&d_B, sizeof(float)*(nnzB));
        hipMalloc((void**)&d_RowB, sizeof(int)*(m+1));
        hipMalloc((void**)&d_ColB, sizeof(int)*(nnzB));


        hipMemcpy(d_A, h_A, nnzA*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_RowA, h_RowA, (m+1)*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_ColA, h_ColA, nnzA*sizeof(int), hipMemcpyHostToDevice);

        hipMemcpy(d_B, h_B, nnzB*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_RowB, h_RowB, (m+1)*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_ColB, h_ColB, nnzB*sizeof(int), hipMemcpyHostToDevice);

        cusparseStatus = hipsparseCreateMatDescr(&descrA);
        hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL);
                hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO);

        cusparseStatus = hipsparseCreateMatDescr(&descrB);
        hipsparseSetMatType(descrB,HIPSPARSE_MATRIX_TYPE_GENERAL);
                hipsparseSetMatIndexBase(descrB,HIPSPARSE_INDEX_BASE_ZERO);

        cusparseStatus = hipsparseCreateMatDescr(&descrC);
        hipsparseSetMatType(descrC,HIPSPARSE_MATRIX_TYPE_GENERAL);
                hipsparseSetMatIndexBase(descrC,HIPSPARSE_INDEX_BASE_ZERO);
        //////////////////////////////////////////////////////////////////////////
        hipMalloc((void**)&d_RowC, sizeof(int)*(m+1));

        hipsparseXcsrgeamNnz(
                handle,
                m,
                n,
                descrA,
                nnzA,
                d_RowA,
                d_ColA,
                descrB,
                nnzB,
                d_RowB,
                d_ColB,
                descrC,
                d_RowC,
                nnzTotalDevHostPtr);

        if (NULL != nnzTotalDevHostPtr)
        {
                nnzC = *nnzTotalDevHostPtr;
        } else {
                hipMemcpy(&nnzC, d_RowC+m, sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&baseC, d_RowC, sizeof(int), hipMemcpyDeviceToHost);
                nnzC -= baseC;
        }


        //////////////////////////////////////////////////////
        //////////////////////////////////////////////////////
        cout << "[nnzC First]" << nnzC << endl;

        vector<int> c_csr_row_ptrs;
        for(int i=0; i<=m; i++)//这里先创建出一个连续存放了m+1个元素的vector,然后将vector转成array
                c_csr_row_ptrs.push_back(-1);

        vector<int> c_cols;
        vector<float> c_vals;
        for(int i=0; i<nnzC; i++) { //这里先创建出一个连续存放了nnzC个元素的vector,然后将vector转成array
                c_cols.push_back(-1);
                c_vals.push_back(-1);
        }
        ///host上的矩阵相乘后的结果
        float *h_C = &c_vals[0];
        int *h_RowC = &c_csr_row_ptrs[0];
        int *h_ColC = &c_cols[0];
        //////////////////////////////////////////////////////
        //////////////////////////////////////////////////////



        hipMalloc((void**)&d_ColC, sizeof(int)*nnzC);
        hipMalloc((void**)&d_C, sizeof(float)*nnzC);


	float alpha = 0.0;
	float beta = 0.0;

        hipsparseScsrgeam(
                handle,
                m,
                n,
                &alpha,////////
                descrA,
                nnzA,
                d_A,
                d_RowA,
                d_ColA,
		&beta,///////
                descrB,
                nnzB,
                d_B,
                d_RowB,
                d_ColB,
                descrC,
                d_C,
                d_RowC,
                d_ColC);


        hipMemcpy(h_C, d_C, nnzC*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(h_RowC, d_RowC, (m+1)*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_ColC, d_ColC, nnzC*sizeof(int), hipMemcpyDeviceToHost);

        hipsparseDestroy(handle);

        hipFree(d_A);
        hipFree(d_RowA);
        hipFree(d_ColA);

        hipFree(d_B);
        hipFree(d_RowB);
        hipFree(d_ColB);

        hipFree(d_C);
        hipFree(d_RowC);
        hipFree(d_ColC);

        hipDeviceReset();

        //////////////////////////////////////////////////////
        //////////////////////////////////////////////////////  
        CSR csr_C(-1);
        csr_C.csr_rows_max = m-1;//csr_rows下标的最大值，若下标最大为3，则实际有4行
        //csr_C.csr_cols_max = k-1;//csr_cols下标的最大值，若下标最大为3，则实际有4列

        cout << "[nnzC]" << nnzC << endl;
        for(int i=0; i<=m; i++) {
        //      cout << "[h_RowC]"<< h_RowC[i] << endl; 
                csr_C.csr_row_ptrs.push_back(h_RowC[i]);
        }
        for(int i=0; i<nnzC; i++) {
        //      cout << "[h_ColC]"<< h_ColC[i] << endl;
                csr_C.csr_cols.push_back(h_ColC[i]);
        }
        for(int i=0; i<nnzC; i++) {
        //      cout << "[h_C]"<< h_C[i] << endl;
                csr_C.csr_vals.push_back(h_C[i]);
        }
        //////////////////////////////////////////////////////
        //////////////////////////////////////////////////////

        return csr_C;
}


CSR useSparseAddition(CSR csr_A, CSR csr_B){


        float* ValA = &csr_A.csr_vals[0];
        int* RowA = &csr_A.csr_row_ptrs[0];
        int* ColA = &csr_A.csr_cols[0];
        //以上代码在给矩阵A赋值，使用CSR格式

        float* ValB = &csr_B.csr_vals[0];
        int* RowB = &csr_B.csr_row_ptrs[0];
        int* ColB = &csr_B.csr_cols[0];
        //以上代码在给矩阵B赋值，使用CSR格式 


        CSR csr_C = SparseAddition(
                csr_A.csr_rows_max+1,//int m,
                csr_A.csr_cols_max+1,//int n,
                csr_A.csr_vals.size(),//int nnzA,
                csr_B.csr_vals.size(),//int nnzB,

                ValA,//float *h_A,
                RowA,//int *h_RowA,
                ColA,//int *h_ColA,

                ValB,//float *h_B,
                RowB,//int *h_RowB,
                ColB //int *h_ColB
           );



        cout << "+++1++++" << endl;
        //for(int i=0; i<= csr_C.csr_rows_max+1; i++) //行数m = 下标最大值csr_rows_max+1
        //      cout << csr_C.csr_row_ptrs[i] << endl;
        cout << "addition csr_c.csr_row_ptrs.size " << csr_C.csr_row_ptrs.size() << endl;

        cout << "+++2++++" << endl;
        //for(int i=0; i< csr_C.csr_cols.size(); i++) 
        //      cout << csr_C.csr_cols[i] << endl;
        cout <<"addition csr_c.csr_cols.size " << csr_C.csr_cols.size() << endl;

        cout << "+++3++++" << endl;
        //for(int i=0; i< csr_C.csr_vals.size(); i++) 
        //      cout << csr_C.csr_vals[i] << endl;    
        cout<< "addition csr_c.csr_vals.size " << csr_C.csr_vals.size() << endl;
        return csr_C;
}
////////////////SparseAddition end/////////////////////////////////////////////////////////////////////////////////////

//////////////////MPI_GPUTaskAssigner begin//////////////////////////////////////////////////////////////////////////////
void getFileInfosInCurrentDir(const char* dir, vector< pair< string, unsigned int > >& fileInfos)
{
	/*/文件句柄  
	long  hFile = 0;
	//文件信息  
	struct _finddata_t fileinfo;
	string p;
	if ((hFile = _findfirst(p.assign(dir).append("\\*").c_str(), &fileinfo)) != -1)
	{
		do
		{
			if (!(fileinfo.attrib &  _A_SUBDIR))
			{
				string filePath = p.assign(dir).append("\\").append(fileinfo.name);
				unsigned long fileSize = fileinfo.size;
				fileInfos.push_back( make_pair(filePath, fileSize) );
			} 
		} while (_findnext(hFile, &fileinfo) == 0);
		_findclose(hFile);
	}*/

	cout << "CurDir"<< dir << endl;	
	
    	struct stat file_stats;
    	DIR *dirp;
    	struct dirent* dent;
    	dirp=opendir(dir); // specify directory here: "." is the "current directory"
   	do {
        	dent = readdir(dirp);
     
   		if (dent) {  
			if(strcmp(dent->d_name,".")==0 || strcmp(dent->d_name,"..")==0)
				continue;

			//extern int errno;
			errno = 0; 

			string cooFile ="/"; 
			cooFile = dir + cooFile + dent->d_name;
	
			stat(cooFile.c_str(), &file_stats);
			if(errno !=0 )
				printf("%s\n",strerror(errno));


			if (!stat(cooFile.c_str(), &file_stats))
			{
				//string fileName = dent->d_name;
				unsigned int fileSize = ( unsigned int )file_stats.st_size;
				//fileInfos.push_back( make_pair(fileName, fileSize) );
 				fileInfos.push_back( make_pair(cooFile, fileSize) );
				cout << "cooFile " << cooFile << " bytes " << fileSize << endl;				
			}
			else
			{
				printf("(stat() failed for this file)\n");
			}
        	}
    	} while (dent);
    	closedir(dirp);	
}


bool Less(const pair<string, unsigned int>& p1, const pair<string, unsigned int>& p2){
	return p1.second < p2.second;
}



void useGetFileInfosInCurrentDir(const char* dir_str, vector< pair< string, unsigned int > >& fileInfos) {
	/*char * dir = "D:\\SparkData";
	vector< pair<string, unsigned long> > fileInfos;
	getFileInfosInCurrentDir(dir, fileInfos);
	for (int i = 0; i<fileInfos.size(); i++)
	{
		cout << "filePath " << fileInfos[i].first << " fileSize " << fileInfos[i].second << endl;
	}*/
	//char* dir = new char[dir_str.length() + 1];
	getFileInfosInCurrentDir(dir_str, fileInfos);

	sort(fileInfos.begin(), fileInfos.end(), Less);
}



///////
void saveAsTaskFile(string fileNameDir, int devCount, vector< pair<string, unsigned int> > fileInfos) {
	
	//目录不为空则删除，然后新建目录
	if(NULL!=opendir(fileNameDir.c_str())){ 
		string cmd = "rm -rf ";
		cmd += fileNameDir.c_str();
		cout << "[cmd]" <<cmd.c_str() <<endl;
		system( cmd.c_str() );
	}
	mkdir(fileNameDir.c_str(), 0775);
	
	
	//int eachDevTaskSize = fileInfos.size() / devCount;

	cout <<" devCount " << devCount <<endl;
	for (int i = 0; i< devCount ; i++) {

		ostringstream oss;//在string后连接int等类型
		oss << fileNameDir << "//" << i << ".txt";
		string fileNameTmp = oss.str();		 

		ofstream fout(fileNameTmp.c_str());

		for (int j = i; j < fileInfos.size(); j = j + devCount){
			cout << fileInfos[j].first << endl;
			fout << fileInfos[j].first << endl;
		}
		fout.close();
	}
}




///////
vector<string> getLocalCOOFiles(string taskTmpDataDir, int dev){
	vector<string> localCOOFiles;

 	ostringstream oss;//在string后连接int等类型
        oss << taskTmpDataDir << "/" << dev << ".txt";
        string taskTmpDataFile = oss.str();


	ifstream fin(taskTmpDataFile.c_str());//在g++中使用字符串str时，要用str.c_str()

        string line;
        while (getline(fin, line))
        {
                cout << "Read from file: " << line << endl;
                localCOOFiles.push_back(line.c_str());
        }


	return localCOOFiles;
}




template < class T >
void ClearVector( vector< T >& vt ) 
{	////swap()是交换函数，使vector离开其自身的作用域，从而强制释放vector所占的内存空间
    	vector< T > vecTemp; 
    	vecTemp.swap( vt );
}


void MPI_GPUTaskAssigner(int argc, char **argv){
	int ierr, num_procs, my_id;


	///////////////////////////////////////////////////////////////////////////////////                             
        int devCount;
        hipGetDeviceCount(&devCount);
        printf("There are %d CUDA devices.\n", devCount);
        ///////////////////////////////////////////////////////////////////////////////////     


	ierr = MPI_Init(&argc, &argv);

	/* find out MY process ID, and how many processes were started. */

	ierr = MPI_Comm_rank(MPI_COMM_WORLD, &my_id);
	ierr = MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

	cout<< "num_procs " << num_procs <<endl;
	
	int namelen;
	char processor_name[MPI_MAX_PROCESSOR_NAME];
	MPI_Get_processor_name(processor_name, &namelen); 	
	printf("out Spawning from %s %d\n", processor_name,my_id);


	///////////////////////////////////////////////////////////////////////////////////
	int dev = (my_id-1)%devCount ;//my_id==0是主进程不干活
	hipSetDevice(dev);	
	//////////////////////////////////////////////////////////////
        string taskTmpDataDir = "/gruntdata/app_data/zhangmeng.zm/TuTongKuang_01265/TmpDataNew";	
	int tag = 0;
	int number;
 	
	if(my_id == 0) {	
	
		//把处理的文件，按大小排序后发送给2个GPU进行均衡分配
		//MPI_Send(&fileNames[0], fileNames.size(), MPI_INT, 1, 0 , MPI_COMM_WORLD);
	

 		MPI_Get_processor_name(processor_name, &namelen);
    		printf("my_id == 0  Spawning from %s \n", processor_name);


		vector< pair<string, unsigned int> > fileInfos;
		useGetFileInfosInCurrentDir("/gruntdata/app_data/zhangmeng.zm/TuTongKuang_01265/COOFilesNew", fileInfos);
	
		for (int i = 0; i<fileInfos.size(); i++)
            		cout << "sortedFilePath " << fileInfos[i].first << " fileSize " << fileInfos[i].second << endl;
        	

		saveAsTaskFile(taskTmpDataDir.c_str(), devCount, fileInfos); 

		////////
        	//把处理的文件，按大小排序后发送给2个GPU进行均衡分配
		number = -1 ; 
		for(int i=1; i<num_procs; i++)
        		MPI_Send(&number, 1, MPI_INT, i, tag , MPI_COMM_WORLD);
	
	} else
	{
		////////
		MPI_Recv(&number, 1, MPI_INT, 0, tag , MPI_COMM_WORLD, MPI_STATUS_IGNORE);

		MPI_Get_processor_name(processor_name, &namelen);
                printf("my_id == %d  Spawning from %s \n", my_id, processor_name);

		///////
		vector<string> localCOOFiles = getLocalCOOFiles(taskTmpDataDir.c_str() , dev);
		printf("my_id == %d  localCOOFiles.size() : %d\n", my_id, localCOOFiles.size() );
		for(int i=0; i<localCOOFiles.size(); i++) {

			///////////////////////////////////////////////////////////////////////////////////		
			cout << "### " << localCOOFiles[i].c_str() << endl;
			clock_t t1 = clock();

			COO coo = useCOOFileReader(localCOOFiles[i].c_str(), ','); //COO文件中分隔符为英文逗号
			clock_t t2 = clock();
			cout << "[useCOOFileReader cost: " << (t2-t1)/CLOCKS_PER_SEC << " seconds]" << endl;
				

			CSR csr = useSparseCOO2CSR(coo);
                        clock_t t3 = clock();
                        cout << "[useSparseCOO2CSR cost: " << (t3-t2)/CLOCKS_PER_SEC << " seconds]" << endl;


			CSC csc = useSparseCSR2CSC(csr);
                        clock_t t4 = clock();
                        cout << "[useSparseCSR2CSC cost: " << (t4-t3)/CLOCKS_PER_SEC << " seconds]" << endl;


			CSR csr_C = useSparseMultiply(csr, csc.use_CSC_Create_CSR());
			//CSR csr_C = useSparseAddition(csr, csr);
			clock_t t5 = clock();
                        cout << "[useSparseMultiply cost: " << (t5-t4)/CLOCKS_PER_SEC << " seconds]" << endl;
			//cout << "[useSparseAddition cost: " << (t5-t4)/CLOCKS_PER_SEC << " seconds]" << endl;
			///////////////////////////////////////////////////////////////////////////////////	
		}

		printf("Hello world! I'm process %i out of %i processes , I am on Dev %i \n", 
				my_id, num_procs, dev);		
	}
	
	ierr = MPI_Finalize();
}
//////////////////MPI_GPUTaskAssigner end//////////////////////////////////////////////////////////////////////////////

//////////////////div_by_row_and_multiply begin////////////////////////////////////////////////////////////////////////
void MPI_GPUTaskAssigner_New(int argc, char **argv){
        int ierr, num_procs, my_id;


        ///////////////////////////////////////////////////////////////////////////////////                             
        int devCount;
        hipGetDeviceCount(&devCount);
        printf("There are %d CUDA devices.\n", devCount);
        ///////////////////////////////////////////////////////////////////////////////////     


        ierr = MPI_Init(&argc, &argv);

        /* find out MY process ID, and how many processes were started. */

        ierr = MPI_Comm_rank(MPI_COMM_WORLD, &my_id);
        ierr = MPI_Comm_size(MPI_COMM_WORLD, &num_procs);

        cout<< "num_procs " << num_procs <<endl;

        int namelen;
        char processor_name[MPI_MAX_PROCESSOR_NAME];
        MPI_Get_processor_name(processor_name, &namelen);
        printf("out Spawning from %s %d\n", processor_name,my_id);


        ///////////////////////////////////////////////////////////////////////////////////
        int dev = (my_id-1)%devCount ;//my_id==0是主进程不干活
        hipSetDevice(dev);
        //////////////////////////////////////////////////////////////
        string taskTmpDataDir = "/gruntdata/app_data/zhangmeng.zm/TuTongKuang_01265/TmpDataNew";
        int tag = 0;
        int number;



	////所有的COO文件及其大小
 	vector< pair<string, unsigned int> > fileInfos;
        useGetFileInfosInCurrentDir("/gruntdata/app_data/zhangmeng.zm/TuTongKuang_01265/COOFilesNew", fileInfos);




        if(my_id == 0) {
                //把处理的文件，按大小排序后，进行均衡分配并写入两个info文件中，然后通知2个GPU去读各自的info文件
                MPI_Get_processor_name(processor_name, &namelen);
                printf("my_id == 0  Spawning from %s \n", processor_name);



                for (int i = 0; i<fileInfos.size(); i++)
                        cout << "sortedFilePath " << fileInfos[i].first << " fileSize " << fileInfos[i].second << endl;


                saveAsTaskFile(taskTmpDataDir.c_str(), devCount, fileInfos);

                ////////
                //把处理的文件，按大小排序后发送给2个GPU进行均衡分配
                number = -1 ;
                for(int i=1; i<num_procs; i++)
                        MPI_Send(&number, 1, MPI_INT, i, tag , MPI_COMM_WORLD);/////这里只是发一个同步操作，并不传实际数据

        } else {
                /////这里只是进行一个同步操作，具体数据在my_id==0的机器的文件系统中
                MPI_Recv(&number, 1, MPI_INT, 0, tag , MPI_COMM_WORLD, MPI_STATUS_IGNORE);

                MPI_Get_processor_name(processor_name, &namelen);
                printf("my_id == %d  Spawning from %s \n", my_id, processor_name);

                ///////vector中元素数量不多时，没必要使用set、hash_set
        	vector<string> localCOOFiles = getLocalCOOFiles(taskTmpDataDir.c_str() , dev); 

		printf("my_id == %d  localCOOFiles.size() : %d\n", my_id, localCOOFiles.size() );
                printf("my_id == %d  fileInfos.size() : %d\n", my_id, fileInfos.size() );

		vector<CSC> csc_all ;
		vector<CSR> csr_part ;
		for(int i=0; i < fileInfos.size(); i++) {
		
                        ///////////////////////////////////////////////////////////////////////////////////             
                        string tmp_all_file = fileInfos[i].first.c_str();
			cout << "[### "<< i << " ] " << tmp_all_file << endl;
                        clock_t t1 = clock();

                        COO coo = useCOOFileReader(tmp_all_file, ','); //COO文件中分隔符为英文逗号
                        clock_t t2 = clock();
                        cout << "[useCOOFileReader cost: " << (t2-t1)/CLOCKS_PER_SEC << " seconds]" << endl;
                                

                        CSR csr = useSparseCOO2CSR(coo);
 			ClearVector(coo.coo_rows);//将COO对象占用的内存空间释放掉
                        ClearVector(coo.coo_cols);
                        ClearVector(coo.coo_vals);
                        clock_t t3 = clock();
			cout << "[useSparseCOO2CSR cost: " << (t3-t2)/CLOCKS_PER_SEC << " seconds]" << endl;


                        if(std::find(localCOOFiles.begin(), localCOOFiles.end(), tmp_all_file) != localCOOFiles.end())
			{	//只将本组需要的csr载入内存
				csr_part.push_back(csr);
				cout << "[csr_part.size]" << csr_part.size() << "[csr_part add]" << tmp_all_file  << endl;
			}

			
                        CSC csc = useSparseCSR2CSC(csr);
                        ClearVector(csr.csr_row_ptrs);//将CSR对象占用的内存空间释放掉
                        ClearVector(csr.csr_cols);
                        ClearVector(csr.csr_vals);
			clock_t t4 = clock();
 			cout << "[useSparseCSR2CSC cost: " << (t4-t3)/CLOCKS_PER_SEC << " seconds]" << endl;

			csc_all.push_back(csc);//需要将所有的csc载入内存
			cout << "[csc_all.size]" << csc_all.size() << endl;
                        ///////////////////////////////////////////////////////////////////////////////////     
                }

		for(int k=0; k<csc_all.size(); k++)
		{
			cout << "[csc_all " << k << "][csc_col_prts.size]" << csc_all[k].csc_col_ptrs.size() << endl;
			cout << "[csc_all " << k << "][csc_rows.size]" << csc_all[k].csc_rows.size() << endl;
			cout << "[csc_all " << k << "][csc_vals.size]" << csc_all[k].csc_vals.size() << endl;
			
			CSR csc_all_csr =  csc_all[k].use_CSC_Create_CSR();
			cout << "[csc_all_csr " << k << "][csr_row_prts.size]" << csc_all_csr.csr_row_ptrs.size() << endl;
			cout << "[csc_all_csr " << k << "][csr_cols.size]" << csc_all_csr.csr_cols.size() << endl;
			cout << "[csc_all_csr " << k << "][csr_vals.size]" << csc_all_csr.csr_vals.size() << endl;
		}

                //////////////////////////////////////////////////////////////////////////////////////
                //////////////////////////////////////////////////////////////////////////////////////
		for(int i=0; i< csr_part.size(); i++){
			for(int j=0; j<csc_all.size(); j++){
		
				///////multiply csr_a csc_b
				clock_t t6 = clock();
                		CSR csr_c = useSparseMultiply(csr_part[i], csc_all[j].use_CSC_Create_CSR());
				clock_t t7 = clock();
				cout << "[csr " << i <<"][left_cols_max"<<csr_part[i].csr_cols_max << "]"<< endl;
				cout << "[csc " << j <<"][right_rows_max"<<csc_all[j].use_CSC_Create_CSR().csr_rows_max<<"]"<<endl;
                		cout << "[useSparseMultiply cost: " << (t7-t6)/CLOCKS_PER_SEC << " seconds]" << endl;
				cout << "==================================================================" << endl;
					
				ClearVector(csr_c.csr_row_ptrs);//将CSR对象占用的内存空间释放掉
                        	ClearVector(csr_c.csr_cols);
                        	ClearVector(csr_c.csr_vals);
			}
			
			ClearVector(csr_part[i].csr_row_ptrs);//将CSR对象占用的内存空间释放掉
                        ClearVector(csr_part[i].csr_cols);
                        ClearVector(csr_part[i].csr_vals);
		}






		/*
                string file_a = "/gruntdata/app_data/zhangmeng.zm/TuTongKuang_01265/COOFilesNew/div_by_row_suffix_mod_11_3";
                string file_b = "/gruntdata/app_data/zhangmeng.zm/TuTongKuang_01265/COOFilesNew/div_by_row_suffix_mod_11_9";

                cout << "### file_a " << file_a << endl;
                cout << "### file_b " << file_b << endl;
                clock_t t1 = clock();


                //////left csr_a
                COO coo_a = useCOOFileReader(file_a, ','); //COO文件中分隔符为英文逗号
                clock_t t2 = clock();
                cout << "[useCOOFileReader coo_a cost: " << (t2-t1)/CLOCKS_PER_SEC << " seconds]" << endl;


                CSR csr_a = useSparseCOO2CSR(coo_a);
                clock_t t3 = clock();
                cout << "[useSparseCOO2CSR csr_a cost: " << (t3-t2)/CLOCKS_PER_SEC << " seconds]" << endl;
                ClearVector(coo_a.coo_rows);//将COO对象占用的内存空间释放掉
                ClearVector(coo_a.coo_cols);
                ClearVector(coo_a.coo_vals);


                ///////right csc_b
                COO coo_b = useCOOFileReader(file_b, ','); //COO文件中分隔符为英文逗号
                clock_t t4 = clock();
                cout << "[useCOOFileReader coo_b cost: " << (t4-t3)/CLOCKS_PER_SEC << " seconds]" << endl;


                CSR csr_b = useSparseCOO2CSR(coo_b);
                clock_t t5 = clock();
                cout << "[useSparseCOO2CSR csr_b cost: " << (t5-t4)/CLOCKS_PER_SEC << " seconds]" << endl;
                ClearVector(coo_b.coo_rows);//将COO对象占用的内存空间释放掉
                ClearVector(coo_b.coo_cols);
                ClearVector(coo_b.coo_vals);


                CSC csc_b = useSparseCSR2CSC(csr_b);
                clock_t t6 = clock();
                cout << "[useSparseCSR2CSC csc_b cost: " << (t6-t5)/CLOCKS_PER_SEC << " seconds]" << endl;


                ///////multiply csr_a csc_b
                CSR csr_C = useSparseMultiply(csr_a, csc_b.use_CSC_Create_CSR());
                clock_t t7 = clock();
                cout << "[useSparseMultiply cost: " << (t7-t6)/CLOCKS_PER_SEC << " seconds]" << endl;*/
                //////////////////////////////////////////////////////////////////////////////////////
                //////////////////////////////////////////////////////////////////////////////////////


                printf("Hello world! I'm process %i out of %i processes , I am on Dev %i \n",
                                my_id, num_procs, dev);
        }

        ierr = MPI_Finalize();
}
//////////////////div_by_row_and_multiply end////////////////////////////////////////////////////////////////////////



int main(int argc, char **argv)
{
	MPI_GPUTaskAssigner_New(argc, argv);
	//MPI_GPUTaskAssigner(argc, argv); 
	return 0;
} 
