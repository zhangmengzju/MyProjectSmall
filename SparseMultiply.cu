#include <hip/hip_runtime.h>
#include <iostream>
#include "hipsparse.h"
using namespace std;

int SparseMultiply(int m,int n, int k,int nnzA,int nnzB,float *h_A,int *h_RowA,int *h_ColA,float *h_B,int *h_RowB,int *h_ColB,float *h_C,int *h_RowC,int *h_ColC)
{
	int baseC,nnzC;
	// nnzTotalDevHostPtr points to host memory
	int *nnzTotalDevHostPtr = &nnzC;

	hipsparseHandle_t handle=0;
	hipsparseStatus_t cusparseStatus;

	hipsparseMatDescr_t descrA=0;
	hipsparseMatDescr_t descrB=0;
	hipsparseMatDescr_t descrC=0;

	float *d_A;
	int *d_RowA;
	int *d_ColA;

	float *d_B;
	int *d_RowB;
	int *d_ColB;

	float *d_C;
	int *d_RowC;
	int *d_ColC;


	cusparseStatus = hipsparseCreate(&handle);

	hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

	hipMalloc((void**)&d_A, sizeof(float)*(nnzA));
	hipMalloc((void**)&d_RowA, sizeof(int)*(m+1));
	hipMalloc((void**)&d_ColA, sizeof(int)*(nnzA));

	hipMalloc((void**)&d_B, sizeof(float)*(nnzB));
	hipMalloc((void**)&d_RowB, sizeof(int)*(k+1));
	hipMalloc((void**)&d_ColB, sizeof(int)*(nnzB));


	hipMemcpy(d_A, h_A, nnzA*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_RowA, h_RowA, (m+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_ColA, h_ColA, nnzA*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(d_B, h_B, nnzB*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_RowB, h_RowB, (k+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_ColB, h_ColB, nnzB*sizeof(int), hipMemcpyHostToDevice);

	cusparseStatus = hipsparseCreateMatDescr(&descrA);
	hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO);

	cusparseStatus = hipsparseCreateMatDescr(&descrB);
	hipsparseSetMatType(descrB,HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatIndexBase(descrB,HIPSPARSE_INDEX_BASE_ZERO);

	cusparseStatus = hipsparseCreateMatDescr(&descrC);
	hipsparseSetMatType(descrC,HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatIndexBase(descrC,HIPSPARSE_INDEX_BASE_ZERO);
	//////////////////////////////////////////////////////////////////////////
	hipMalloc((void**)&d_RowC, sizeof(int)*(m+1));

	hipsparseXcsrgemmNnz(
		handle,
		HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		m,
		n,
		k,
		descrA,
		nnzA,
		d_RowA,
		d_ColA,
		descrB,
		nnzB,
		d_RowB,
		d_ColB,
		descrC,
		d_RowC,
		nnzTotalDevHostPtr);

	if (NULL != nnzTotalDevHostPtr)
	{
		nnzC = *nnzTotalDevHostPtr;
	} else {
		hipMemcpy(&nnzC, d_RowC+m, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&baseC, d_RowC, sizeof(int), hipMemcpyDeviceToHost);
		nnzC -= baseC;
	}

	
	hipMalloc((void**)&d_ColC, sizeof(int)*nnzC);
	hipMalloc((void**)&d_C, sizeof(float)*nnzC);

	
	hipsparseScsrgemm(
		handle, 
		HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		HIPSPARSE_OPERATION_NON_TRANSPOSE, 
		m, 
		n, 
		k,
		descrA, 
		nnzA,
		d_A, 
		d_RowA, 
		d_ColA,
		descrB,
		nnzB,
		d_B, 
		d_RowB, 
		d_ColB,
		descrC,
		d_C, 
		d_RowC, 
		d_ColC);


	hipMemcpy(h_C, d_C, nnzC*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_RowC, d_RowC, (m+1)*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_ColC, d_ColC, nnzC*sizeof(int), hipMemcpyDeviceToHost);

	hipsparseDestroy(handle);

	hipFree(d_A);
	hipFree(d_RowA);
	hipFree(d_ColA);

	hipFree(d_B);
	hipFree(d_RowB);
	hipFree(d_ColB);

	hipFree(d_C);
	hipFree(d_RowC);
	hipFree(d_ColC);

	hipDeviceReset();
	return 1;
}

int main()
{
	float* ValA = new float[9];
    int* RowA = new int[5];
    int* ColA = new int[9]; 

    float* pValA = ValA;
    int* pRowA = RowA;
    int* pColA = ColA;

    *pValA = 1.0F; pValA++;
    *pValA = 4.0F; pValA++;
    *pValA = 2.0F; pValA++;
	*pValA = 3.0F; pValA++;
	*pValA = 5.0F; pValA++;
	*pValA = 7.0F; pValA++;
	*pValA = 8.0F; pValA++;
	*pValA = 9.0F; pValA++;
	*pValA = 6.0F; pValA++;

	*pRowA = 0; pRowA++;
	*pRowA = 2; pRowA++;
	*pRowA = 4; pRowA++;
	*pRowA = 7; pRowA++;
	*pRowA = 9; pRowA++;

	*pColA = 0; pColA++;
	*pColA = 1; pColA++;
	*pColA = 1; pColA++;
	*pColA = 2; pColA++;
	*pColA = 0; pColA++;
	*pColA = 3; pColA++;
	*pColA = 4; pColA++;
	*pColA = 2; pColA++;
	*pColA = 4; pColA++;
	//以上代码在给矩阵A赋值，使用CSR格式
	
	float* ValB = new float[5];
	int* RowB = new int[6];
	int* ColB = new int[5]; 

	float* pValB = ValB;
	int* pRowB = RowB;
	int* pColB = ColB;

	*pValB = 1.0F; pValB++;
	*pValB = 2.0F; pValB++;
	*pValB = 3.0F; pValB++;
	*pValB = 4.0F; pValB++;
	*pValB = 5.0F; pValB++;

	*pRowB = 0; pRowB++;
	*pRowB = 1; pRowB++;
	*pRowB = 2; pRowB++;
	*pRowB = 3; pRowB++;
	*pRowB = 4; pRowB++;
	*pRowB = 5; pRowB++;
	//以上代码在给矩阵B赋值，使用CSR格式
	
	*pColB = 0; pColB++;
	*pColB = 1; pColB++;
	*pColB = 2; pColB++;
	*pColB = 0; pColB++;
	*pColB = 1; pColB++;

	float* ValC = new float[20];
	int* RowC = new int[10];
	int* ColC = new int[20]; 

	int c = SparseMultiply(
		4,// int m,
		3,//int n,
		5,//int k,
		9,//int nnzA,
		5,//int nnzB,
		ValA,//float *h_A,
		RowA,// int *h_RowA,
		ColA,//int *h_ColA,
		ValB,// float *h_B,
		RowB,//  int *h_RowB,
		ColB,// int *h_ColB,
		ValC,//float *h_C,
		RowC,//int *h_RowC,
		ColC//int *h_ColC)
	   );
	   
	cout<< "SparseMultiply Result" << c << endl;  
	   
	return 0;
} 
