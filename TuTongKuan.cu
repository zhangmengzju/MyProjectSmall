include <stdio.h>
#include <cstdlib>
#include <algorithm>
#include <iostream>
#include <fstream>
#include <string>
#include <deque>
#include <vector>
#include <iostream>
#include <omp.h>
#include <mpi.h>
#include <stdlib.h>
#include <sstream>

#include <hip/hip_runtime.h>
#include <cstring>

#include "caffe/caffe.hpp"
#include "caffe/util/io.hpp"
#include "caffe/blob.hpp"

#include <tfs.h>
#include <ImageReader.h>
#include <CSVReader.h>

using namespace std;
using namespace caffe;

#define BUFSIZE 256
#define TAG 0
void MPI_init(int argc, char** argv, int &device_id, int& myid, int& numprocs)
{
  int devCount;
  char idstr[256];
  char idstr2[256];
  char buff[BUFSIZE];
  int i;
  int rank, namelen;
  char processor_name[MPI_MAX_PROCESSOR_NAME];

  /**
   * for Mvapich2
   */
//      rank = atoi(getenv("MV2_COMM_WORLD_RANK"));
  /**
   * for OpenMPI
   */
  //MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  rank = atoi(getenv("OMPI_COMM_WORLD_LOCAL_RANK"));
  myid = rank;

  hipGetDeviceCount(&devCount);
  device_id = myid % devCount;
  hipSetDevice(device_id);
  printf("rank=%d, devCount=%d, device_id=%d\n", rank, devCount, device_id);
  
  PI_Status stat;
  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &numprocs);

  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  myid = rank;
  MPI_Get_processor_name(processor_name, &namelen);

  if (myid == 0)
  {
    printf("  We have %d processors\n", numprocs);
    printf("  Spawning from %s \n", processor_name);
    printf("  CUDA MPI\n");
    printf("\n");
    for (i = 1; i < numprocs; i++)
    {
      buff[0] = 'I';
      MPI_Send(buff, BUFSIZE, MPI_CHAR, i, TAG, MPI_COMM_WORLD);
    }

    //hipGetDeviceCount(&devCount);
    //device_id = myid % devCount;
    buff[1] = '\0';
    idstr[0] = '\0';
    if (devCount == 0)
    {
      sprintf(idstr, "- %-11s %5d %4d NONE", processor_name, rank, devCount);
    }
    else
    {
      if (devCount >= 1)
      {
        sprintf(idstr, "+ %-11s %5d %4d", processor_name, rank, devCount);
        idstr2[0] = '\0';
        //        for (int i = 0; i < devCount; ++i)
        {

          hipDeviceProp_t devProp;
          hipGetDeviceProperties(&devProp, device_id);
          sprintf(idstr2, " %s (%d) ", devProp.name, device_id);
          strncat(idstr, idstr2, BUFSIZE);
        }
      }
      else
      {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        sprintf(idstr, "%-11s %5d %4d %s", processor_name, rank, devCount, devProp.name);
      }
    }
    strncat(buff, idstr, BUFSIZE);
    
    printf("  Probing nodes...\n");
    printf("     Node        Psid  CUDA Cards (devID)\n");
    printf("     ----------- ----- ---- ----------\n");

    printf("%s\n", buff);

    for (i = 1; i < numprocs; i++)
    {
      MPI_Recv(buff, BUFSIZE, MPI_CHAR, i, TAG, MPI_COMM_WORLD, &stat);
      printf("%s\n", buff);
    }
    printf("\n");
    //    MPI_Finalize();

  }
  else
  {
    MPI_Recv(buff, BUFSIZE, MPI_CHAR, 0, TAG, MPI_COMM_WORLD, &stat);
    MPI_Get_processor_name(processor_name, &namelen);
    buff[1] = '\0';
    idstr[0] = '\0';
    if (devCount == 0)
    {
      sprintf(idstr, "- %-11s %5d %4d NONE", processor_name, rank, devCount);
    }
    else
    {
      if (devCount >= 1)
      {
        sprintf(idstr, "+ %-11s %5d %4d", processor_name, rank, devCount);
        idstr2[0] = '\0';

        //        for (int i = 0; i < devCount; ++i)
        {
          hipDeviceProp_t devProp;
          hipGetDeviceProperties(&devProp, device_id);
          sprintf(idstr2, " %s (%d) ", devProp.name, device_id);
          strncat(idstr, idstr2, BUFSIZE);
        }
      }
      else
      {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, device_id);
        sprintf(idstr, "%-11s %5d %4d %s", processor_name, rank, devCount, devProp.name);
      }
    }
    strncat(buff, idstr, BUFSIZE);
    MPI_Send(buff, BUFSIZE, MPI_CHAR, 0, TAG, MPI_COMM_WORLD);
  }

}

int main(int argc, char **argv)
{
  if (argc != 3)
  {
    LOG(ERROR) << "TuTongKuan TableName1 TableName2";
    return 1;
  }
  int device_id;
  int rank_id;
  int np;
  MPI_init(argc, argv, device_id, rank_id, np);
  char buf[128];
  FILE *pp;
  char cmd[] = "dship download ";

  if ((pp = popen(cmd, "r")) == NULL)
  {
    printf("popen() error!/n");
    exit(1);
  }

  while (fgets(buf, sizeof buf, pp))
  {
    printf("%s", buf);
  }
  pclose(pp);

  MPI_Finalize();
  return 0;
}